#include "hip/hip_runtime.h"
/*
* Implement gradient descent part of autograd engine in CUDA
*
* Author: Andrew Boessen
*/

#include <stddef.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <hiprand/hiprand_kernel.h>

extern "C" {
#include "engine.h"
}

extern "C" {
void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}

/**
 * This helper function allocates new memory for a specified amout of Values.
 *
 * @param v (return paramter) The pointer to the start of the Values in memory
 * @param num Number of values to allocate
 */
void allocValue(Value* v, size_t num) {
    hipError_t err = hipMallocManaged(&v, num * sizeof(Value));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMallocManaged failed while allocating Value: %s\n", hipGetErrorString(err));
        // Handle the error appropriately
        exit(1);
    }
    // Prefetch memory to correct devices (e.g. CPU or GPU)
    hipMemPrefetchAsync(v, num * sizeof(Value), MAIN_DEVICE);
}

/**
 * This helper function allocates new memory for an array of Values.
 *
 * @param prt (return parameter) Pointer to start of list of Values
 * @param len Length of array of Value
 */
void allocValueArr(Value** ptr, size_t len) {
    hipError_t err = hipMallocManaged(&ptr, len * sizeof(Value*));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMallocManaged failed while allocating array of Values: %s\n", hipGetErrorString(err));
        // Handle the error appropriately
        exit(1);
    }
    // Prefetch memory to correct devices (e.g. CPU or GPU)
    hipMemPrefetchAsync(ptr, len * sizeof(Value*), MAIN_DEVICE);

}

/**
 * Function to calculate gradient of Value object that is a sum
 * 
 * Computes gradient with respect to the operands
 *
 * @param v Pointer to the Value object resulting from addition
 */
__device__ void add_backwards(Value* v) {
    v->children[0]->grad += v->grad;
    v->children[1]->grad += v->grad;
}

/**
 * Function to calculate gradient of Value object that is a difference
 *
 * Computes the gradient with respect to the operands
 *
 * @param v Pointer to Value object resulting from subtraction
 *
 * @note
 * The final gradient for the operand is its local gradient multiplied by any external gradient flowing from a parent.
 * The local derivative for the subtraction is:
 *     dv/da (locally) = 1
 *     dv/db (locally) = -1
 * The external gradient (from parent nodes) is stored in v->grad.
 * Thus, the final gradient for a is: dv/da = 1 * v->grad
 * And for b is: dv/db = -1 * v->grad
 */
__device__ void sub_backwards(Value* v) {
    v->children[0]->grad += v->grad;
    v->children[1]->grad -= v->grad;
}

/**
 * Computes the gradient of the multiplication operation with respect to its operands.
 *
 * @param v Pointer to the Value object resulting from the multiplication.
 *
 * @note
 * The final gradient for the operand is its local gradient multiplied by any external gradient flowing from a parent.
 * The local derivative for the multiplication is:
 *     dv/da (locally) = b
 *     dv/db (locally) = a
 * The external gradient (from parent nodes) is stored in v->grad.
 * Thus, the final gradient for a is: dv/da = b * v->grad
 * And for b is: dv/db = a * v->grad
 */
__device__ void mul_backward(Value* v) {
    // printf("child %.f grad = %f*%f", v->children[0], v->children[1]->val, v->grad);
    // printf("child %.f grad = %f*%f", v->children[1], v->children[0]->val, v->grad);
    v->children[0]->grad += v->children[1]->val * v->grad;
    v->children[1]->grad += v->children[0]->val * v->grad;
}

/**
 * Computes the gradient of the division operation with respect to its operands.
 *
 * @param v Pointer to the Value object resulting from the division.
 *
 * @note
 * The final gradient for the operand is its local gradient multiplied by any external gradient flowing from a parent.
 * The local derivative for the division is:
 *     dv/da (locally) = 1/b
 *     dv/db (locally) = -a/(b^2)
 * The external gradient (from parent nodes) is stored in v->grad.
 * Thus, the final gradient for a is: dv/da = (1/b) * v->grad
 * And for b is: dv/db = (-a/(b^2)) * v->grad
 */
__device__ void div_backward(Value* v) {
    v->children[0]->grad += (1.0 / v->children[1]->val) * v->grad;
    v->children[1]->grad += (-v->children[0]->val / (v->children[1]->val * v->children[1]->val)) * v->grad;
}

/**
 * Computes the gradient of the power operation with respect to its operands.
 *
 * @param v Pointer to the Value object resulting from the power operation.
 *
 * @note
 * The final gradient for the operand is its local gradient multiplied by any external gradient flowing from a parent.
 * The local derivative for the power operation is:
 *     dv/da (locally) = b * a^(b-1)
 *     dv/db (locally) = a^b * log(a)
 * The external gradient (from parent nodes) is stored in v->grad.
 * Thus, the final gradient for a is: dv/da = (b * a^(b-1)) * v->grad
 * And for b is: dv/db = (v * log(a)) * v->grad
 */
__device__ void power_backward(Value* v) {
    v->children[0]->grad += (v->children[1]->val * pow(v->children[0]->val, v->children[1]->val - 1)) * v->grad;
    if (v->children[0]->val > 0) {  // Ensure base is positive before computing log
        v->children[1]->grad += (log(v->children[0]->val) * pow(v->children[0]->val, v->children[1]->val)) * v->grad;
    }
}

/**
 * This helper function doubles the capacity of array. It does this by allocating a new
 * array with double the capacity, copying the existing data to the new array,
 * and updating the topo pointer to point to the new array.
 *
 * @param arr Pointer to the pointer that holds the array.
 * @param arr_size Pointer to the variable that holds the current size of the array.
 * @param arr_capacity Pointer to the variable that holds the current capacity of the array.
 */
void resize_array(Value*** arr, int* arr_size, int* arr_capacity) {
    *arr_capacity *= 2;
    Value** new_arr = (Value**)realloc(*arr, *arr_capacity * sizeof(Value*));
    if (new_arr == NULL) {
        printf("Memory allocation failed.\n");
        exit(1);
    }
    *arr = new_arr;
}

/**
 * Helper function to build a topological order of the computation graph, starting from the given Value object.
 *
 * @param v The starting Value object for the topological sort.
 * @param topo A pointer to an array where the topological order will be stored.
 * @param topo_size Pointer to the size of the topo array.
 * @param visited Pointer to an array that keeps track of visited Value objects.
 * @param visited_size Pointer to the size of the visited array.
 */
void build_topo(Value* v, Value*** topo, int* topo_size, int* topo_capacity, Value*** visited, int* visited_size, int* visited_capacity) {
    for (int i = 0; i < *visited_size; ++i) {
        if ((*visited)[i] == v) return;
    }

    if (*visited_size == *visited_capacity) {
        resize_array(visited, visited_size, visited_capacity);
    }
    (*visited)[*visited_size] = v;
    (*visited_size)++;

    for (int i = 0; i < v->n_children; ++i) {
        build_topo(v->children[i], topo, topo_size, topo_capacity, visited, visited_size, visited_capacity);
    }

    if (*topo_size == *topo_capacity) {
        resize_array(topo, topo_size, topo_capacity);
    }
    (*topo)[*topo_size] = v;
    (*topo_size)++;
}
}