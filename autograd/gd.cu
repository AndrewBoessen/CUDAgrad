#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <hiprand/hiprand_kernel.h>

extern "C" {
#include "engine.h"
}

extern "C" {
void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}

/**
 * Function to create a list of nodes in topological order in CUDA memory
 *
 * @param root
 */
Value* create_topological_list_cuda(Value* root) {
    int num_nodes = 0;
    int num_visited = 0;
    Value** visited = (Value**)malloc(INITIAL_SIZE * sizeof(Value*));
    Value** sorted_nodes = (Value**)malloc(INITIAL_SIZE * sizeof(Value*));
    int topo_capacity = INITIAL_SIZE;
    int visited_capacity = INITIAL_SIZE;

    build_topo(root, &sorted_nodes, &num_nodes, &topo_capacity, &visited, &num_visited, &visited_capacity);

    // Allocate memory on the GPU
    Value* cuda_nodes;
    hipMalloc((void**)&cuda_nodes, num_nodes * sizeof(Value));

    // Copy data from CPU to GPU
    hipMemcpy(cuda_nodes, sorted_nodes, num_nodes * sizeof(Value), hipMemcpyHostToDevice);

    // Free CPU memory
    free(sorted_nodes);
    free(visited);

    return cuda_nodes;
}


/**
 * Function to calculate gradient of Value object that is a sum
 * 
 * Computes gradient with respect to the operands
 *
 * @param v Pointer to the Value object resulting from addition
 */
__device__ void add_backwards(Value* v) {
    v->children[0]->grad += v->grad;
    v->children[1]->grad += v->grad;
}

/**
 * Function to calculate gradient of Value object that is a difference
 *
 * Computes the gradient with respect to the operands
 *
 * @param v Pointer to Value object resulting from subtraction
 *
 * @note
 * The final gradient for the operand is its local gradient multiplied by any external gradient flowing from a parent.
 * The local derivative for the subtraction is:
 *     dv/da (locally) = 1
 *     dv/db (locally) = -1
 * The external gradient (from parent nodes) is stored in v->grad.
 * Thus, the final gradient for a is: dv/da = 1 * v->grad
 * And for b is: dv/db = -1 * v->grad
 */
__device__ void sub_backwards(Value* v) {
    v->children[0]->grad += v->grad;
    v->children[1]->grad -= v->grad;
}

/**
 * Computes the gradient of the multiplication operation with respect to its operands.
 *
 * @param v Pointer to the Value object resulting from the multiplication.
 *
 * @note
 * The final gradient for the operand is its local gradient multiplied by any external gradient flowing from a parent.
 * The local derivative for the multiplication is:
 *     dv/da (locally) = b
 *     dv/db (locally) = a
 * The external gradient (from parent nodes) is stored in v->grad.
 * Thus, the final gradient for a is: dv/da = b * v->grad
 * And for b is: dv/db = a * v->grad
 */
__device__ void mul_backward(Value* v) {
    // printf("child %.f grad = %f*%f", v->children[0], v->children[1]->val, v->grad);
    // printf("child %.f grad = %f*%f", v->children[1], v->children[0]->val, v->grad);
    v->children[0]->grad += v->children[1]->val * v->grad;
    v->children[1]->grad += v->children[0]->val * v->grad;
}

/**
 * Computes the gradient of the division operation with respect to its operands.
 *
 * @param v Pointer to the Value object resulting from the division.
 *
 * @note
 * The final gradient for the operand is its local gradient multiplied by any external gradient flowing from a parent.
 * The local derivative for the division is:
 *     dv/da (locally) = 1/b
 *     dv/db (locally) = -a/(b^2)
 * The external gradient (from parent nodes) is stored in v->grad.
 * Thus, the final gradient for a is: dv/da = (1/b) * v->grad
 * And for b is: dv/db = (-a/(b^2)) * v->grad
 */
__device__ void div_backward(Value* v) {
    v->children[0]->grad += (1.0 / v->children[1]->val) * v->grad;
    v->children[1]->grad += (-v->children[0]->val / (v->children[1]->val * v->children[1]->val)) * v->grad;
}

/**
 * Computes the gradient of the power operation with respect to its operands.
 *
 * @param v Pointer to the Value object resulting from the power operation.
 *
 * @note
 * The final gradient for the operand is its local gradient multiplied by any external gradient flowing from a parent.
 * The local derivative for the power operation is:
 *     dv/da (locally) = b * a^(b-1)
 *     dv/db (locally) = a^b * log(a)
 * The external gradient (from parent nodes) is stored in v->grad.
 * Thus, the final gradient for a is: dv/da = (b * a^(b-1)) * v->grad
 * And for b is: dv/db = (v * log(a)) * v->grad
 */
__device__ void power_backward(Value* v) {
    v->children[0]->grad += (v->children[1]->val * pow(v->children[0]->val, v->children[1]->val - 1)) * v->grad;
    if (v->children[0]->val > 0) {  // Ensure base is positive before computing log
        v->children[1]->grad += (log(v->children[0]->val) * pow(v->children[0]->val, v->children[1]->val)) * v->grad;
    }
}
}