#include "hip/hip_runtime.h"
/*
* Implementation of nueral network lib with GPU acceleration
*
* Author: Andrew Boessen
*/

#include <stddef.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <hiprand/hiprand_kernel.h>

extern "C" {
#include "nn.h"
#include "engine.h"
}

extern "C"{
__device__ void mul_dev(Value* w, Value* x, Value* v) {
    v->val = w->val * x->val;
    v->grad = 0;
    v->children[0] = w;
    v->children[1] = x;
    v->n_children = 2;
    v->op = MUL;
}

__device__ void add_dev(Value* out, Value* b, Value* v) {
    v->val = out->val + b->val;
    v->grad = 0;
    v->children[0] = out;
    v->children[1] = b;
    v->n_children = 2;
    v->op = ADD;
}

__device__ void relu_dev(Value* out, Value* v) {
    v->val = (out->val < 0) ? 0 : out->val;
    v->grad = 0;
    v->children[0] = out;
    v->n_children = 1;
    v->op = RELU;
}

/**
 * This helper function allocates new memory for a specified amount of Neurons.
 *
 * @param n (return parameter) The pointer to the start of the Neurons in memory
 */
void allocNeuron(Neuron** n) {
    hipError_t err = hipMalloc(n, sizeof(Neuron));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMallocManaged failed while allocating Neuron: %s\n", hipGetErrorString(err));
        // Handle the error appropriately
        exit(1);
    }
}

/**
 * This helper function allocates new memory for an array of Neurons.
 *
 * @param ptr (return parameter) Pointer to start of list of Neurons
 * @param len Length of array of Neurons
 */
void allocNeuronArr(Neuron*** ptr, size_t len) {
    hipError_t err = hipMalloc(ptr, len * sizeof(Neuron*));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMallocManaged failed while allocating array of Neurons: %s\n", hipGetErrorString(err));
        // Handle the error appropriately
        exit(1);
    }
}

/**
 * @brief Initialize a neuron with random weights and zero bias.
 *
 * @param nin Number of input connections.
 * @param nonlin Activation function flag (1 for ReLU, 0 for linear).
 * @return Pointer to the initialized Neuron.
 */
Neuron* init_neuron(int nin, int nonlin) {
    Neuron* neuron;
    allocNeuron(&neuron);
    hipMalloc(&(neuron->w), nin);
    for (int i = 0; i < nin; i++) {
        // random values between -1 and 1
        hipMemcpy(neuron->w[i], init_value((rand() % 2000 - 1000) / 1000.0), sizeof(Value), hipMemcpyHostToDevice);
    }
    hipMemcpy(&neuron->b, init_value(0), sizeof(Value), hipMemcpyHostToDevice);
    hipMemcpy(&neuron->nin, &nin, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(&neuron->nonlin, &nonlin, sizeof(int), hipMemcpyHostToDevice);;
    return neuron;
}

/**
 * @brief Initialize a neural network layer with specified neurons.
 *
 * @param nin Number of input connections for each neuron.
 * @param nout Number of neurons in the layer.
 * @param nonlin Activation function flag for all neurons (1 for ReLU, 0 for linear).
 * @return Pointer to the initialized Layer.
 */
Layer* init_layer(int nin, int nout, int nonlin) {
    Layer* layer;
    // Allocate one layer in memory
    hipMalloc(&layer, sizeof(Layer));
    // Allocate neurons that make up layer
    allocNeuronArr(&(layer->neurons), nout);
    for (int i = 0; i < nout; i++) {
        // Init neurons
        hipMemcpy(layer->neurons[i], init_neuron(nin, nonlin), sizeof(Neuron), hipMemcpyDeviceToDevice);
    }
    hipMemcpy(&layer->nout, &nout, sizeof(int), hipMemcpyHostToDevice);
    return layer;
}

/**
 * @brief Perform forward pass computation for a layer.
 *
 * @param layer Pointer to the layer.
 * @param x Array of input values for the layer.
 * @param out Array of values corresponding to output of layer.
 * @param products Array of Values for products of inputs and weights.
 * @param biases Values to store sum of output ands bias
 * @param activations Values to store outputs activation function
 */
__global__ void layer_forward(Layer* layer, Value** x, Value** out, Value** products, Value** biases, Value** activations) {
    // Id of datapoint in batch
    int datapoint_id = blockIdx.y;

    // Index of neuron to computer (block)
    int neuron_idx = blockIdx.x;
    // Current neuron in layer
    Neuron* n = layer->neurons[neuron_idx];

    // Index of cuurent input of neuron
    int input_idx = blockDim.x * blockIdx.y + threadIdx.x;

    // Index of product within array
    int prod_idx = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;

    // Index of nuerons output
    int out_idx = datapoint_id * gridDim.x + neuron_idx;

    // Set paramters of product
    Value* prod = products[prod_idx];
    mul_dev(n->w[threadIdx.x], x[input_idx], prod);

    // Add product to children of neuron output
    out[out_idx]->children[threadIdx.x] = prod;
    // Update neuron output value
    atomicAdd(&(out[out_idx]->val), prod->val);

    // Wait for all thread to finish computing products
    __syncthreads();

    // Add bias to sum and activate if nonlin
    // Only run if last thread in block
    if (threadIdx.x == blockDim.x - 1) {
        Value* sum = biases[out_idx];
        add_dev(out[out_idx], n->b, sum);
        
        out[out_idx] = sum;

        if (n->nonlin) {
            // Activate with ReLU function if nonlin
            Value* relu_val = activations[out_idx];
            relu_dev(out[out_idx], relu_val);

            out[out_idx] = relu_val;
        }
    }
}

/**
 * @brief Initialize a Multilayer Perceptron (MLP) with the specified layer sizes.
 *
 * @param sizes Array of layer sizes, where each element represents the number of neurons in that layer.
 * @param nlayers Number of layers in the MLP.
 * @return Pointer to the initialized MLP.
 */
MLP* init_mlp(int* sizes, int nlayers) {
    // Allocate memory for MLP
    MLP* mlp;
    hipMalloc(&mlp, sizeof(MLP));
    // Allocate space for layers in MLP
    hipMalloc(&(mlp->layers), (nlayers - 1) * sizeof(Layer*));
    for (int i = 0; i < nlayers - 1; i++) {
        int nonlin = (i != nlayers - 2);  // nonlinearity for all layers except the last one
        mlp->layers[i] = init_layer(sizes[i], sizes[i+1], nonlin);
    }
    mlp->nlayers = nlayers - 1;
    return mlp;
}

/**
 * @brief Perform forward pass computation for the entire MLP.
 *
 * @param mlp Pointer to the MLP.
 * @param x Array of input values for the MLP.
 * @param nin Number of inputs
 * @return Array of output values from the final layer of the MLP.
 */
Value** mlp_forward(MLP* mlp, Value** in, int nin) {
    // Copy input array to device
    Value** x;
    hipMalloc(&x, nin * sizeof(Value*));
    for (int i = 0; i < nin; i++) {
        hipMalloc(&x[i], sizeof(Value));
        hipMemcpy(&x[i], in[i], sizeof(Value), hipMemcpyDeviceToHost);
    }

    for (int i = 0; i < mlp->nlayers; i++) {
        Layer* curr_layer = mlp->layers[i];

        int curr_layer_out;
        hipMemcpy(&curr_layer_out, &curr_layer->nout, sizeof(int), hipMemcpyDeviceToHost);

        Value** out;
        hipMalloc(&out, curr_layer_out * sizeof(Value*));
        for(int i = 0; i < curr_layer_out; i++) {
            hipMalloc(&out[i]->children, nin * sizeof(Value*));
        }

        Value** products;
        hipMalloc(&products, nin * curr_layer_out * sizeof(Value*));
        for(int i = 0; i < nin * curr_layer_out; i++) {
            hipMalloc(&out[i]->children, 2 * sizeof(Value*));
        }

        // Grid size: single datapoint so y is 1
        dim3 grid_size(curr_layer->nout, 1);
        layer_forward<<<grid_size, nin>>>(curr_layer, x, out, products, out, out);
        // Wait for kernel to finish before updating x
        hipDeviceSynchronize();
        // Number of next inputs are number of current outputs
        nin = curr_layer->nout;
        // Next layers inputs are current layers outputs
        x = out;
    }
    
    // Copy outputs to host
    Value** out= (Value**)malloc(nin * sizeof(Value*));
    for (int i = 0; i < nin; i++) {
        hipMemcpy(&out[i], &x[i], sizeof(Value), hipMemcpyDeviceToHost);
    }
    return out;
}

/**
 * @brief Helper function to free arrays of allocated Value in maanged memory
 *
 * @param arr Array of Value arrs to free
 */
void freePtrArr(Value*** arrs, int len) {
    for (int i = 0; i < len; i++) {
        Value** curr_arr = arrs[i];
        // Loop until NULL pointer encountered
        for (int j = 0; curr_arr[j] != NULL; j++) {
            Value* curr_val = curr_arr[j];
            hipError_t err_c = hipFree(curr_val->children);
            hipError_t err = hipFree(curr_val);
        }
        hipFree(curr_arr);
    } 
}

/**
 * @brief Train the MLP for one batch
 *
 * Do a forward pass for an entire batch of data points,
 * then do a backward pass to find gradients and update paramters
 *
 * @param mlp MLP object to train
 * @param x inputs for the batch
 * @param nin number of neurons in input layer
 * @param y_true ground truth for datapoints in batch
 * @param lr learning rate
 * @param batch_size number of datapoints in batch
 * @return Total loss of entire batch
 */
/*
float train(MLP* mlp, Value** x, int nin, Value** y_true, float lr, int batch_size){
    // Arrays for storing Value arrays to later be freed
    Value** products_ptrs[mlp->nlayers];
    Value** bias_ptrs[mlp->nlayers];
    Value** act_ptrs[mlp->nlayers];
    Value** sum_ptrs[mlp->nlayers];

    for (int l = 0; l < mlp->nlayers; l++) {
        Layer* curr_layer = mlp->layers[l];
        // Total number of neurons in entire batch
        int total_neurons = curr_layer->nout * batch_size;

        // Grid dimensions: x for neurons in layer, y for batch size
        dim3 grid_size(curr_layer->nout, batch_size);
        layer_forward<<<grid_size, nin>>>(curr_layer, x, out, products, biases, activations);
        // Wait for kernel to finish before updating x
        hipDeviceSynchronize();
        // Number of next inputs are number of current outputs
        nin = curr_layer->nout;
        // Next layers inputs are current layers outputs
        x = out;

        // Add Value arrs to arrays to free
        products_ptrs[l] = products;
        bias_ptrs[l] = biases;
        act_ptrs[l] = activations;
        sum_ptrs[l] = sums;
    }
    // Calculate loss for each output
    Value* total_loss = init_value(0.0);

    for (int i = 0; i < batch_size * nin; i+=nin) {
        Value* curr_data_out[nin];
        Value* curr_data_gt[nin];
        // Populate array with slice from output
        for (int j = 0; j < nin; j++) {
            curr_data_out[j] = x[i + j];
            curr_data_gt[j] = y_true[i + j];
        }
        // Calculate loss for each datapoint in batch
        Value* loss = mse_loss(curr_data_out, curr_data_gt, nin);

        // Add datapoint loss to total loss
        total_loss = add(total_loss, loss);
    }
    // Do backprop to find gradients
    backward(total_loss);
    // Single step for batch
    update_weights(mlp, lr);
    // zero grads before next batch
    zero_grad(mlp);
    
    // Free network from memory
    freePtrArr(sum_ptrs, mlp->nlayers);
    freePtrArr(products_ptrs, mlp->nlayers);
    freePtrArr(bias_ptrs, mlp->nlayers);
    freePtrArr(act_ptrs, mlp->nlayers);

    return total_loss->val;
}
*/

/**
 * @brief Compute the mean squared error (MSE) loss between predicted and true values.
 *
 * @param y_pred Array of predicted values.
 * @param y_true Array of true values.
 * @param size Number of values in y_pred and y_true arrays.
 * @return Pointer to the computed MSE loss value.
 */
Value* mse_loss(Value** y_pred, Value** y_true, int size) {
    
    Value* loss = init_value(0.0);
    for (int i = 0; i < size; i++) {
        Value* diff = sub(y_pred[i], y_true[i]);
        Value* sq = power(diff, init_value(2.0));
        loss = add(loss, sq);
    }
    loss = divide(loss, init_value(size));

    return loss;
}

/**
 * @brief CUDA kernel to zero weight and bias of mlp
 *
 * @param layers Layers of the MLP to update
 */
__global__ void zero_grad_kernel(Layer** layers) {
    // Id for layer
    int layer_idx = blockIdx.x;
    Layer* l = layers[layer_idx];

    // Id for neuron within id
    int neuron_idx = blockIdx.y % l->nout;
    Neuron* n = l->neurons[neuron_idx];

    // Get param to update from thread id
    int weight_idx = threadIdx.x % n->nin;
    // Zero grads for weight and bias
    n->w[weight_idx]->grad = 0;
    n->b->grad = 0;
    
}

/**
 * @brief Host function to zero gradients for params in MLP
 *
 * @param mlp MLP to zero grads
 */
void zero_grad(MLP* mlp) {
    // Get maximum layer size
    int max_neurons = 0;
    for (int i = 0; i < mlp->nlayers; i++) {
        if (mlp->layers[i]->nout > max_neurons) {
            max_neurons = mlp->layers[i]->nout;
        }
    }
    // Dimensions of grid
    // X dim is for layers
    // Y dim is for neurons in layer
    dim3 grid_size(mlp->nlayers, max_neurons);
    // Call Kernel to zero params' grads
    zero_grad_kernel<<<grid_size,max_neurons>>>(mlp->layers);
    // Wait for kernel to finish
    hipDeviceSynchronize();
}

/**
 * @brief Device func to update the weights of a value using gradient descent.
 *
 * @param v Pointer to the value whose weights need to be updated.
 * @param lr Learning rate for the weight update.
 */
__device__ __inline__ void update_weights_dev(Value* v, float lr) {
    v->val -= lr * v->grad;
}

/**
 * @brief CUDA kernel to update paramaters of MLP
 *
 * @param layers Layers of the MLP to update
 * @param lr Learning rate
 */
__global__ void update_params(Layer** layers, float lr) {
    // Id for layer
    int layer_idx = blockIdx.x;
    Layer* l = layers[layer_idx];

    // Id for neuron within id
    int neuron_idx = blockIdx.y;
    if (neuron_idx <= l->nout - 1) {
        Neuron* n = l->neurons[neuron_idx];

        // Get param to update from thread id
        int weight_idx = threadIdx.x;

        if (weight_idx <= n->nin - 1) {
            update_weights_dev(n->w[weight_idx], lr);
        }
        if(weight_idx == n->nin -1) {
            update_weights_dev(n->b, lr);
        }
    }
}

/**
 * @brief Host function to update weight
 *
 * @param mlp MLP to update weights for
 */
void update_weights(MLP* mlp, float lr) {
    // Get maximum layer size
    int max_neurons = 0;
    for (int i = 0; i < mlp->nlayers; i++) {
        if (mlp->layers[i]->nout > max_neurons) {
            max_neurons = mlp->layers[i]->nout;
        }
    }
    // Dimensions of grid
    // X dim is for layers
    // Y dim is for neurons in layer
    dim3 grid_size(mlp->nlayers, max_neurons);

    update_params<<<grid_size, max_neurons>>>(mlp->layers, lr);

    hipDeviceSynchronize();
}

/**
 * @brief Display the parameters (weights and biases) of the MLP.
 *
 * @param mlp Pointer to the MLP.
 */
void show_params(MLP* mlp){
    printf("\nMLP\n");
    for (int i = 0; i < mlp->nlayers; i++) {
        Layer* layer = mlp->layers[i];
        printf("\nLayer%i:\n", i);
        for (int j = 0; j < layer->nout; j++) {
            Neuron* neuron = layer->neurons[j];
            for (int k = 0; k < neuron->nin; k++) {
                print_value(neuron->w[k]);
            }
        }
    }
    printf("\n\n");
}

/**
 * @brief Free the memory allocated for a neuron.
 *
 * @param neuron Pointer to the neuron to be freed.
 */
void free_neuron(Neuron* neuron) {
    for (int i = 0; i < neuron->nin; i++) {
        free_value(neuron->w[i]);
    }
    hipFree(neuron->w);
    free_value(neuron->b);
    hipFree(neuron);
}

/**
 * @brief Free the memory allocated for a layer.
 *
 * @param layer Pointer to the layer to be freed.
 */
void free_layer(Layer* layer) {
    for (int i = 0; i < layer->nout; i++) {
        free_neuron(layer->neurons[i]);
    }
    hipFree(layer->neurons);
    hipFree(layer);
}

/**
 * @brief Free the memory allocated for the entire MLP.
 *
 * @param mlp Pointer to the MLP to be freed.
 */
void free_mlp(MLP* mlp) {
    for (int i = 0; i < mlp->nlayers; i++) {
        free_layer(mlp->layers[i]);
    }
    hipFree(mlp->layers);
    hipFree(mlp);
}
}
