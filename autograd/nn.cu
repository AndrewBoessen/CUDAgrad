#include "hip/hip_runtime.h"
/*
* Implementation of nueral network lib with GPU acceleration
*
* Author: Andrew Boessen
*/
#include <stddef.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <hiprand/hiprand_kernel.h>

extern "C" {
#include "nn.h"
#include "engine.h"
}

extern "C"{
__device__ void mul_dev(Value* w, Value* x, Value* v) {
    v->grad = 0;
    v->children[0] = w;
    v->children[1] = x;
    v->n_children = 2;
     v->val = w->val * x->val;
    v->op = MUL;
}

__device__ void add_dev(Value* out, Value* b, Value* v) {
    v->val = out->val + b->val;
    v->grad = 0;
    v->children[0] = out;
    v->children[1] = b;
    v->n_children = 2;
    v->op = ADD;
}

__device__ void relu_dev(Value* out, Value* v) {
    v->val = (out->val < 0) ? 0 : out->val;
    v->grad = 0;
    v->children[0] = out;
    v->n_children = 1;
    v->op = RELU;
}

/**
 * This helper function allocates new memory for a specified amount of Neurons.
 *
 * @param n (return parameter) The pointer to the start of the Neurons in memory
 * @param num Number of neurons to allocate
 */
void allocNeuron(Neuron** n, size_t num) {
    hipError_t err = hipMallocManaged(n, num * sizeof(Neuron));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMallocManaged failed while allocating Neuron: %s\n", hipGetErrorString(err));
        // Handle the error appropriately
        exit(1);
    }
}

/**
 * This helper function allocates new memory for an array of Neurons.
 *
 * @param ptr (return parameter) Pointer to start of list of Neurons
 * @param len Length of array of Neurons
 */
void allocNeuronArr(Neuron*** ptr, size_t len) {
    hipError_t err = hipMallocManaged(ptr, len * sizeof(Neuron*));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMallocManaged failed while allocating array of Neurons: %s\n", hipGetErrorString(err));
        // Handle the error appropriately
        exit(1);
    }
}

/**
 * @brief Initialize a neuron with random weights and zero bias.
 *
 * @param nin Number of input connections.
 * @param nonlin Activation function flag (1 for ReLU, 0 for linear).
 * @return Pointer to the initialized Neuron.
 */
Neuron* init_neuron(int nin, int nonlin) {
    Neuron* neuron;
    allocNeuron(&neuron, 1);
    allocValueArr(&(neuron->w), nin);
    for (int i = 0; i < nin; i++) {
        neuron->w[i] = init_value((rand() % 2000 - 1000) / 1000.0);  // random values between -1 and 1
    }
    neuron->b = init_value(0);
    neuron->nin = nin;
    neuron->nonlin = nonlin;
    return neuron;
}

/**
 * @brief Initialize a neural network layer with specified neurons.
 *
 * @param nin Number of input connections for each neuron.
 * @param nout Number of neurons in the layer.
 * @param nonlin Activation function flag for all neurons (1 for ReLU, 0 for linear).
 * @return Pointer to the initialized Layer.
 */
Layer* init_layer(int nin, int nout, int nonlin) {
    Layer* layer;
    // Allocate one layer in memory
    hipMallocManaged(&layer, sizeof(Layer));
    // Allocate neurons that make up layer
    allocNeuronArr(&(layer->neurons), nout);
    for (int i = 0; i < nout; i++) {
        // Init neurons
        layer->neurons[i] = init_neuron(nin, nonlin);
    }
    layer->nout = nout;
    return layer;
}

/**
 * @brief Perform forward pass computation for a layer.
 *
 * @param layer Pointer to the layer.
 * @param x Array of input values for the layer.
 * @param out Array of values corresponding to output of layer.
 * @param products Array of Values for products of inputs and weights.
 * @param biases Values to store sum of output ands bias
 * @param activations Values to store outputs activation function
 */
__global__ void layer_forward(Layer* layer, Value** x, Value** out, Value** products, Value** biases, Value** activations) {
    // Index of neuron to computer (block)
    int neuron_idx = blockIdx.x;

    // Current neuron in layer
    Neuron* n = layer->neurons[neuron_idx];

    // Index of cuurent input of neuron
    int input_idx = threadIdx.x % blockDim.x;

    // Index of product within array
    int prod_idx = input_idx + blockIdx.x * blockDim.x;

    // Set paramters of product
    Value* prod = products[prod_idx];
    mul_dev(n->w[input_idx], x[input_idx], prod);

    // Add product to children of neuron output
    out[neuron_idx]->children[input_idx] = prod;
    // Update neuron output value
    atomicAdd(&(out[neuron_idx]->val), prod->val);

    // Wait for all thread to finish computing products
    __syncthreads();

    // Add bias to sum and activate if nonlin
    // Only run if last thread in block
    if (input_idx == blockDim.x - 1) {
        Value* sum = biases[neuron_idx];
        add_dev(out[neuron_idx], n->b, sum);
        
        out[neuron_idx] = sum;

        if (n->nonlin) {
            // Activate with ReLU function if nonlin
            Value* relu_val = activations[neuron_idx];
            relu_dev(out[neuron_idx], relu_val);

            out[neuron_idx] = relu_val;
        }
    }
    
}

/**
 * @brief Initialize a Multilayer Perceptron (MLP) with the specified layer sizes.
 *
 * @param sizes Array of layer sizes, where each element represents the number of neurons in that layer.
 * @param nlayers Number of layers in the MLP.
 * @return Pointer to the initialized MLP.
 */
MLP* init_mlp(int* sizes, int nlayers) {
    // Allocate memory for MLP
    MLP* mlp;
    hipMallocManaged(&mlp, sizeof(MLP));
    // Allocate space for layers in MLP
    hipMallocManaged(&(mlp->layers), (nlayers - 1) * sizeof(Layer*));
    for (int i = 0; i < nlayers - 1; i++) {
        int nonlin = (i != nlayers - 2);  // nonlinearity for all layers except the last one
        mlp->layers[i] = init_layer(sizes[i], sizes[i+1], nonlin);
    }
    mlp->nlayers = nlayers - 1;
    return mlp;
}

/**
 * @brief Perform forward pass computation for the entire MLP.
 *
 * @param mlp Pointer to the MLP.
 * @param x Array of input values for the MLP.
 * @param nin Number of inputs
 * @return Array of output values from the final layer of the MLP.
 */
Value** mlp_forward(MLP* mlp, Value** x, int nin) {
    for (int i = 0; i < mlp->nlayers; i++) {
        Layer* curr_layer = mlp->layers[i];

        // Allocate empty value arr for outputs
        float initialSums[curr_layer->nout];
        memset(initialSums, 0.0, curr_layer->nout * sizeof(float));
        // Initialize sums to 0.0
        Value** out = init_values(initialSums, curr_layer->nout);
        // Allocate space for children of outputs
        for(int i = 0; i < curr_layer->nout; i++) {
            allocValueArr(&(out[i]->children), nin);
            out[i]->n_children = nin;
            out[i]->op = ADD;
        }

        // Allocate array for prodcuts of inputs and weights
        Value** products;
        allocValueArr(&products, nin * curr_layer->nout);
        // Allocate space for products children
        for(int i = 0; i < nin * curr_layer->nout; i++) {
            products[i] = init_value(0);
            allocValueArr(&(products[i]->children), 2);
        }

        // Allocate Values to store sum of output ands bias
        Value** biases;
        allocValueArr(&biases, curr_layer->nout);
        for(int i = 0; i < curr_layer->nout; i++) {
            biases[i] = init_value(0);
            allocValueArr(&(biases[i]->children), 2);
        }

        // Allocate Value to store outputs activation function
        Value** activations;
        allocValueArr(&activations, curr_layer->nout);
        for(int i = 0; i < curr_layer->nout; i++) {
            activations[i] = init_value(0);
            allocValueArr(&(activations[i]->children), 1);
        }

        layer_forward<<<curr_layer->nout, nin * curr_layer->nout>>>(curr_layer, x, out, products, biases, activations);
        // Wait for kernel to finish before updating x
        hipDeviceSynchronize();
        // Number of next inputs are number of current outputs
        nin = curr_layer->nout;
        // Next layers inputs are current layers outputs
        x = out;
    }
    return x;
}

/**
 * @brief Compute the mean squared error (MSE) loss between predicted and true values.
 *
 * @param y_pred Array of predicted values.
 * @param y_true Array of true values.
 * @param size Number of values in y_pred and y_true arrays.
 * @return Pointer to the computed MSE loss value.
 */
Value* mse_loss(Value** y_pred, Value** y_true, int size) {
    
    Value* loss = init_value(0.0);
    for (int i = 0; i < size; i++) {
        Value* diff = sub(y_pred[i], y_true[i]);
        Value* sq = power(diff, init_value(2.0));
        loss = add(loss, sq);
    }
    loss = divide(loss, init_value(size));

    return loss;
}

/**
 * @brief Update the weights of a value using gradient descent.
 *
 * @param v Pointer to the value whose weights need to be updated.
 * @param lr Learning rate for the weight update.
 */
__device__ void update_weights(Value* v, float lr) {
    v->val -= lr * v->grad;
}

/**
 * @brief Display the parameters (weights and biases) of the MLP.
 *
 * @param mlp Pointer to the MLP.
 */
void show_params(MLP* mlp){
    printf("\nMLP\n");
    for (int i = 0; i < mlp->nlayers; i++) {
        Layer* layer = mlp->layers[i];
        printf("\nLayer%i:\n", i);
        for (int j = 0; j < layer->nout; j++) {
            Neuron* neuron = layer->neurons[j];
            for (int k = 0; k < neuron->nin; k++) {
                print_value(neuron->w[k]);
            }
        }
    }
        printf("\n\n");
}

/**
 * @brief Free the memory allocated for a neuron.
 *
 * @param neuron Pointer to the neuron to be freed.
 */
void free_neuron(Neuron* neuron) {
    for (int i = 0; i < neuron->nin; i++) {
        free_value(neuron->w[i]);
    }
    hipFree(neuron->w);
    free_value(neuron->b);
    hipFree(neuron);
}

/**
 * @brief Free the memory allocated for a layer.
 *
 * @param layer Pointer to the layer to be freed.
 */
void free_layer(Layer* layer) {
    for (int i = 0; i < layer->nout; i++) {
        free_neuron(layer->neurons[i]);
    }
    hipFree(layer->neurons);
    hipFree(layer);
}

/**
 * @brief Free the memory allocated for the entire MLP.
 *
 * @param mlp Pointer to the MLP to be freed.
 */
void free_mlp(MLP* mlp) {
    for (int i = 0; i < mlp->nlayers; i++) {
        free_layer(mlp->layers[i]);
    }
    hipFree(mlp->layers);
    hipFree(mlp);
}
}
